#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <vector>
#include <fstream>
#include <cstring>
#include <time.h>

#include <hip/hip_runtime.h>
#include <algorithm>
#include <hiprand.h>


#define N 9
#define n 3
#define size 9

__device__ void printtable(int* table)
{
	printf("-------------------------------\n");
	for (int i = 0; i < size; i++)
	{
		for (int j = 0; j < size; j++)
		{
			printf("%d ", table[i * size + j]);
		}
		printf("\n");
	}

	printf("-------------------------------\n");
}

__device__ bool checkEntire(const int* table)
{
	int occur[9];
	for (int i = 0; i < 9; i++) occur[i] = 0;

	//check row
	for (int i = 0; i < 9; i++)
	{
		for (int h = 0; h < 9; h++) occur[h] = 0;
		for (int j = 0; j < 9; j++)
		{
			int val = table[i * 9 + j];
			if (val != 0)
			{
				if (occur[val - 1] == 1)
				{
					return false;
				}
				else
				{
					occur[val - 1] = 1;
				}
			}
		}
	}

	//check column
	for (int i = 0; i < 9; i++)
	{
		for (int h = 0; h < 9; h++) occur[h] = 0;
		for (int j = 0; j < 9; j++)
		{
			int val = table[j * 9 + i];
			if (val != 0)
			{
				if (occur[val - 1] == 1)
				{
					return false;
				}
				else
				{
					occur[val - 1] = 1;
				}
			}
		}
	}

	//check box
	for (int a = 0; a < 3; a++)
	{
		for (int b = 0; b < 3; b++)
		{
			for (int h = 0; h < 9; h++) occur[h] = 0;

			for (int i = 0; i < 3; i++)
			{
				for (int j = 0; j < 3; j++)
				{
					int val = table[(a * 3 + i) * 9 + (b * 3 + j)];
					if (val != 0)
					{
						if (occur[val - 1] == 1)
						{
							return false;
						}
						else
						{
							occur[val - 1] = 1;
						}
					}
				}
			}
		}
	}

	return true;

}

__device__ bool validBoard(const int* board, int changed) {

	int r = changed / 9;
	int c = changed % 9;

	// if changed is less than 0, then just default case
	if (changed < 0) {
		return checkEntire(board);
	}

	if ((board[changed] < 1) || (board[changed] > 9)) {
		return false;
	}

	bool seen[N];
	for (int h = 0; h < 9; h++) seen[h] = false;


	// check if row is valid
	for (int i = 0; i < N; i++) {
		int val = board[r * N + i];

		if (val != 0) {
			if (seen[val - 1]) {
				return false;
			}
			else {
				seen[val - 1] = true;
			}
		}
	}

	// check if column is valid
	for (int h = 0; h < 9; h++) seen[h] = false;
	for (int j = 0; j < N; j++) {
		int val = board[j * N + c];

		if (val != 0) {
			if (seen[val - 1]) {
				return false;
			}
			else {
				seen[val - 1] = true;
			}
		}
	}

	// finally check if the sub-board is valid
	int ridx = r / n;
	int cidx = c / n;

	for (int h = 0; h < 9; h++) seen[h] = false;
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			int val = board[(ridx * n + i) * N + (cidx * n + j)];

			if (val != 0) {
				if (seen[val - 1]) {
					return false;
				}
				else {
					seen[val - 1] = true;
				}
			}
		}
	}

	// if we get here, then the board is valid
	return true;
}

__device__ bool checkValid(int* table, int row, int col, int val)
{
	//check along row, column and cube for the validation of this assignment
	//the start index of coresponding cube 
	for (int h = 0; h < 9; h++)
	{
		//check along row
		if (table[row * size + h] == val)
		{
			return false;
		}
	}

	for (int h = 0; h < 9; h++)
	{
		//check along column
		if (table[h * size + col] == val)
		{
			return false;
		}
	}

	//check within a cube
	int i = row - row % 3;
	int j = col - col % 3;
	for (int h = 0; h < 3; h++)
	{
		for (int k = 0; k < 3; k++)
		{
			int idx_r = i + h;
			int idx_c = j + k;
			if (table[idx_r * size + idx_c] == val) return false;
		}
	}

	return true;
}

bool validChecking(int* table, int row, int col, int val)
{
	//check along row, column and cube for the validation of this assignment
	//the start index of coresponding cube 
	for (int h = 0; h < 9; h++)
	{
		//check along row
		if (table[row * size + h] == val)
		{
			return false;
		}
	}

	for (int h = 0; h < 9; h++)
	{
		//check along column
		if (table[h * size + col] == val)
		{
			return false;
		}
	}

	//check within a cube
	int i = row - row % 3;
	int j = col - col % 3;
	for (int h = 0; h < 3; h++)
	{
		for (int k = 0; k < 3; k++)
		{
			int idx_r = i + h;
			int idx_c = j + k;
			if (table[idx_r * size + idx_c] == val) return false;
		}
	}

	return true;
}

int initial_search(int* new_table, int* old_table)
{
	//generating more possible tables for BFS using
	//find a empty spot in the initial table
	bool found = false;
	int i = 0;
	int count = 0;
	while (i < size * size && !found)
	{
		
		if (old_table[i] == 0)
		{
			found = true;
			//find a valid value for the board
			for (int num = 1; num <= 9; num++)
			{
				int r = i / size;
				int c = i % size;
				if (validChecking(old_table, r, c, num))
				{
					count++;
					//copy to new_board
					for (int j = 0; j < size * size; j++)
					{
						new_table[count * size * size + j] = old_table[j];
					}
					new_table[count * size * size + i] = num;
				}
			}
		}
	}
	return count;
}


void loadToFile(const char* outFile, int* table)
{
	FILE* out = fopen(outFile, "w");
	if (out == NULL)
	{
		printf("Could not open file\n!");
		return;
	}


	fprintf(out, "---------------------------------\n");
	for (int i = 0; i < size; i++)
	{
		for (int j = 0; j < size; j++)
		{
			fprintf(out, "%d ", table[i * size + j]);
		}
		fprintf(out, "\n");
	}
	printf("Solution has loaded to file\n");

}



__global__ void cudaBFS(int* old_table, int* new_table, int total_tables, int* table_index, int* empty_space, int* empty_space_count)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	while (index < total_tables)
	{
		int cur_position = index * size * size;
		int* cur_table = old_table + cur_position;

		//find the next empty spot
		bool found = false;
		int i = 0;

		//find an empty spot
		while ((i < size * size) && (!found))
		{
			if (cur_table[i] == 0)
			{
				found = true;
				//row, column of this position
				int row = i / size;
				int col = i % size;

				//find out the numbers that work at this position
				for (int num = 1; num <= 9; num++)
				{
					if (checkValid(cur_table, row, col, num))
					{
						//copy the board with the empty box filled with num
						//int newindex = *table_index;
						//atomicAdd(table_index, 1);
						int newindex = atomicAdd(table_index, 1);

						//count the number of empty spaces in the new_table
						int count = 0;
						//global memory---should not modify!!!!
						//cur_table[i] = num;
						for (int h = 0; h < size; h++)
						{
							for (int k = 0; k < size; k++)
							{
								new_table[newindex * size * size + h * size + k] = cur_table[h * size + k];
								//record the position of the empty space and the total empty spaces in this table
								if (cur_table[h * size + k] == 0)
								{
									if (h != row || k != col)
									{
										empty_space[newindex * size * size + count] = h * size + k;
										count++;
									}
									else if(h==row && k==col)
									{
										new_table[newindex * size * size + h * size + k] = num;
									}

								}
								


							}
						}
						//cur_table[i] = 0;
						empty_space_count[newindex] = count;
					}
				}
			}
			i++;
		}

		index += blockDim.x * gridDim.x;
	}

}

__device__ bool checkValid(int* table, int pos, int val)
{
	if (pos < 0) return checkEntire(table);
	if (val < 1 || val>9) return false;
	int row = pos / size;
	int col = pos % size;
	//check along row, column and cube for the validation of this assignment
	//the start index of coresponding cube 
	for (int h = 0; h < 9; h++)
	{
		//check along row
		if (h != col && table[row * size + h] == val)
		{
			return false;
		}
	}

	for (int h = 0; h < 9; h++)
	{
		//check along column
		if (h != row && table[h * size + col] == val)
		{
			return false;
		}
	}

	//check within a cube
	int i = row - row % 3;
	int j = col - col % 3;
	for (int h = 0; h < 3; h++)
	{
		for (int k = 0; k < 3; k++)
		{
			int idx_r = i + h;
			int idx_c = j + k;
			if (idx_r != row && idx_c != col && table[idx_r * size + idx_c] == val) return false;
		}
	}

	return true;
}

__global__ void sudokuBacktracking(int* table, const int possible_table_counting, int* empty_space, int* empty_space_count, int* finished, int* result_table)
{

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	int* cur_table;
	int* cur_table_empty;
	int num_empty;
	//if the table still has empty spots, keep filling in numbers
	while (((*finished==0)) && (index < possible_table_counting))
	{
		//index of empty space recorded in cur_empty array
		int empty_index = 0;
		cur_table = table + index * size * size;
		cur_table_empty = empty_space + index * size * size;
		num_empty = empty_space_count[index];


		//empty_index = backtracking(cur_table, cur_table_empty, num_empty, empty_index);
		while ((empty_index >= 0) && (empty_index < num_empty))
		{
			cur_table[cur_table_empty[empty_index]]++;

			if (!validBoard(cur_table, cur_table_empty[empty_index])) {
				//validBoard(cur_table, cur_table_empty[empty_index])
				//!checkValid(cur_table, cur_table_empty[empty_index], cur_table[cur_table_empty[empty_index]])
				// if the board is invalid and we tried all numbers here already, backtrack
				// otherwise continue (it will just try the next number in the next iteration)
				if (cur_table[cur_table_empty[empty_index]] >= 9) {
					cur_table[cur_table_empty[empty_index]] = 0;
					empty_index--;
				}
			}
			// if valid board, move forward in algorithm
			else {
				empty_index++;
			}
		}

		//printf("empty_index:%d, num_empty:%d\n", empty_index, num_empty);

		if (empty_index == num_empty)
		{
			*finished =1;
			for (int i = 0; i < size * size; i++)
			{
				result_table[i] = cur_table[i];
			}

			//printf("-------find solution------");
		}

		//*finished = true;
		//for (int i = 0; i < size * size; i++)
		//{
		//	result_table[i] = cur_table[i];
		//}

		index += blockDim.x * gridDim.x;
	}
}

void loadTable(const char* fileName, int* table)
{
	FILE* in = fopen(fileName, "r");
	if (in == NULL)
	{
		printf("File load fail!");
		return;
	}

	char temp;
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			fscanf(in, "%c\n", &temp);
			if (temp >= '1' && temp <= '9') table[i * N + j] = (int)(temp - '0');
			else table[i * N + j] = 0;
		}
	}

	//in.close();
}

void printTable(int* table)
{
	printf("-------------------------------\n");
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			printf("%d ", table[i * N + j]);
		}
		printf("\n");
	}

	printf("-------------------------------\n");
}

int main(int argc, char* argv[]) {



	const unsigned int threadsPerBlock = atoi(argv[1]);
	const unsigned int maxBlocks = atoi(argv[2]);
	// filename of the starting board
	//const char* filename = "input.txt";
	char* filename = argv[3];
	const char* outfile="result.txt";
	dim3 dimgrid(maxBlocks);
	dim3 dimblock(threadsPerBlock);
	// load the board
	int* board = new int[N * N];
	loadTable(filename, board);

	//alternating used, to store the last round boards and new board generated from last round boards
	int* new_boards;
	int* old_boards;

	//store index of empty space: row*size+col
	int* empty_spaces;
	// stores the number of empty spaces in each board
	int* empty_space_count;
	int* board_index;

	int total_boards = 1;

	//flag for whether solved
	//once equals true: store the table to gpu_solved
	int* finished;
	hipMalloc(&finished, sizeof(int));
	hipMemset(finished, 0, sizeof(int));

	//store a solved sudoku table 
	int* gpu_solved;
	hipMalloc(&gpu_solved, N * N * sizeof(int));
	hipMemcpy(gpu_solved, board, N * N * sizeof(int), hipMemcpyHostToDevice);

	//get copied form gpu_solved
	int* solved = new int[N * N];
	memset(solved, 0, N * N * sizeof(int));


	// the size of memory allocation 
	// a experimental number, numbers of new tables will not exceed sk/81
	const int sk = pow(2, 26);

	int host_count;

	//may affect the execution time
	//differnet sudoku puzzle may have differnet optimum execution time 
	// number of iterations to run BFS for
	int iterations =10;

	// allocate memory on the gpu
	hipMalloc(&empty_spaces, sk * sizeof(int));
	hipMalloc(&empty_space_count, (sk / 81 + 1) * sizeof(int));
	hipMalloc(&new_boards, sk * sizeof(int));
	hipMalloc(&old_boards, sk * sizeof(int));
	hipMalloc(&board_index, sizeof(int));

	// initialize memory
	hipMemset(board_index, 0, sizeof(int));
	hipMemset(new_boards, 0, sk * sizeof(int));
	hipMemset(old_boards, 0, sk * sizeof(int));

	// copy the initial board to the old boards
	hipMemcpy(old_boards, board, N * N * sizeof(int), hipMemcpyHostToDevice);


	struct timespec start, stop;
	double time;
	if (clock_gettime(CLOCK_REALTIME, &start) == -1) { perror("clock gettime"); }

	//----------------------------debugging--------------------------------------------
	//printf("total table: %d", possible_table_counting);
	//int* firstbfs;
	//int* temp;
	//hipMalloc(&temp, possible_table_counting*size*size * sizeof(int));
	//hipMemset(temp, 0, possible_table_counting * size * size * sizeof(int));
	//copytempresult << <dimgrid, dimblock >> > (temp, new_table, possible_table_counting);

	//hipMemcpy(firstbfs, temp, possible_table_counting *size * size * sizeof(int), hipMemcpyDeviceToHost);
	//for (int i = 0; i < possible_table_counting * size * size; i++)
	//{
	//	if (i % 9 == 0)
	//	{
	//		printf("\n");
	//	}
	//	if (i % 81 == 0)
	//	{
	//		printf("---------------------------");
	//	}
	//	printf("%d ", firstbfs[i]);
	//}

	//-----------------------------debugging end----------------------------------------------


	// generate more tables based on the initial boards
	//initial_search(int* new_table, int* old_table)
	cudaBFS << <dimgrid, dimblock >> > (old_boards, new_boards, total_boards, board_index, empty_spaces, empty_space_count);

	// loop through BFS iterations to generate more boards deeper in the tree
	for (int i = 0; i < iterations; i++) {
		hipMemcpy(&host_count, board_index, sizeof(int), hipMemcpyDeviceToHost);

		printf("total boards after an iteration %d: %d\n", i, host_count);

		hipMemset(board_index, 0, sizeof(int));


		if (i % 2 == 0) {
			cudaBFS << <dimgrid, dimblock >> > (new_boards, old_boards, host_count, board_index, empty_spaces, empty_space_count);

		}
		else {
			cudaBFS << <dimgrid, dimblock >> > (old_boards, new_boards, host_count, board_index, empty_spaces, empty_space_count);
		}
	}

	hipMemcpy(&host_count, board_index, sizeof(int), hipMemcpyDeviceToHost);
	printf("new number of boards retrieved is %d\n", host_count);


	if (iterations % 2 == 1) {
		// if odd number of iterations run, then send it old boards not new boards;
		new_boards = old_boards;
	}

	sudokuBacktracking << <dimgrid, dimblock >> > (new_boards, host_count, empty_spaces, empty_space_count, finished, gpu_solved);
	
	// copy back the solved board

	hipMemcpy(solved, gpu_solved, N * N * sizeof(int), hipMemcpyDeviceToHost);


	if (clock_gettime(CLOCK_REALTIME, &stop) == -1) { perror("clock gettime"); }
	time = (stop.tv_sec - start.tv_sec) + (double)(stop.tv_nsec - start.tv_nsec) / 1e9;
	printf("time is %f s\n", time );


	printTable(solved);

	loadToFile(outfile, solved);

	// free memory
	delete[] board;
	delete[] solved;

	hipFree(empty_spaces);
	hipFree(empty_space_count);
	hipFree(new_boards);
	hipFree(old_boards);
	hipFree(board_index);

	hipFree(finished);
	hipFree(gpu_solved);

	return 0;

}
